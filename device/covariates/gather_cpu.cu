#include "hip/hip_runtime.h"
/*
 * Firepony
 *
 * Copyright (c) 2014-2015, NVIDIA CORPORATION
 * Copyright (c) 2015, Nuno Subtil <subtil@gmail.com>
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the copyright holders nor the names of its
 *      contributors may be used to endorse or promote products derived from
 *      this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../../types.h"
#include "../alignment_data_device.h"
#include "../firepony_context.h"
#include "covariates.h"
#include "../expected_error.h"
#include "../empirical_quality.h"

#include "packer_context.h"
#include "packer_cycle_illumina.h"
#include "packer_quality_score.h"
#include "generate_event_key.h"
#include "high_quality_window.h"
#include "gather.h"

#include "../primitives/util.h"

#include "../../table_formatter.h"

#include <thrust/functional.h>

namespace firepony {

// updates a set of covariate tables for a given event
template <typename packer_table, typename... packer_chain>
static void covariate_gatherer(const uint32 tid,
                               firepony_context<host>& ctx, const alignment_batch_device<host>& batch, const uint32 cigar_event_index, packer_table& first, packer_chain&... next)
{
    auto& table = first.table;

    covariate_key_set keys;
    bool key_valid;

    key_valid = generate_covariate_event_key<host, typename packer_table::packer>(keys, ctx, batch, cigar_event_index);

    if (key_valid)
    {
        const uint32 read_index = ctx.cigar.cigar_event_read_index[cigar_event_index];
        const auto idx = batch.crq_index(read_index);
        const auto read_bp_offset = ctx.cigar.cigar_event_read_coordinates[cigar_event_index];

        auto& val_M = table.value(tid, keys.M);
        val_M.observations++;
        val_M.mismatches += ctx.fractional_error.snp_errors[idx.qual_start + read_bp_offset];

        auto& val_I = table.value(tid, keys.I);
        val_I.observations++;
        val_I.mismatches += ctx.fractional_error.insertion_errors[idx.qual_start + read_bp_offset];

        auto& val_D = table.value(tid, keys.D);
        val_D.observations++;
        val_D.mismatches += ctx.fractional_error.deletion_errors[idx.qual_start + read_bp_offset];
    }

    // recurse for next argument
    covariate_gatherer<packer_chain...>(tid, ctx, batch, cigar_event_index, next...);
}

// terminator for covariate_gatherer
template <typename... packer_chain>
static void covariate_gatherer(const uint32, firepony_context<host>&, const alignment_batch_device<host>&, const uint32)
{ }

struct covariate_gather_worker : public lambda<host>
{
    LAMBDA_INHERIT_SYS(host);

    void operator() (const uint32 tid)
    {
        auto& cv = ctx.covariates;

        auto t_qual = make_packer_table<covariate_packer_quality_score<host>>(cv.quality);
        auto t_cycle = make_packer_table<covariate_packer_cycle_illumina<host>>(cv.cycle);
        auto t_context = make_packer_table<covariate_packer_context<host>>(cv.context);

        const uint32 num_threads = command_line_options.cpu_threads;
        constexpr uint32 grain_size = 1000;

        for(uint32 start = grain_size * tid;
            start < ctx.cigar.cigar_event_read_coordinates.size();
            start += grain_size * num_threads)
        {
            const uint32 stop = std::min(start + grain_size, ctx.cigar.cigar_event_read_coordinates.size());

            for(uint32 cigar_event_index = start; cigar_event_index < stop; cigar_event_index++)
            {
                covariate_gatherer(tid, ctx, batch, cigar_event_index,
                                   t_qual, t_cycle, t_context);
            }
        }
    }
};

template <>
void gather_covariates<host>(firepony_context<host>& context, const alignment_batch<host>& batch)
{
    auto& cv = context.covariates;

    // compute the "high quality" windows (i.e., clip off low quality ends from each read)
    cv.high_quality_window.resize(batch.device.num_reads);
    parallel<host>::for_each(context.active_read_list.begin(),
                             context.active_read_list.end(),
                             compute_high_quality_windows<host>(context, batch.device));

    cv.quality.init();
    cv.cycle.init();
    cv.context.init();

    parallel<host>::for_each(command_line_options.cpu_threads,
                             covariate_gather_worker(context, batch.device));
}

} // namespace firepony
