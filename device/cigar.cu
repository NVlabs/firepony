#include "hip/hip_runtime.h"
/*
 * Firepony
 *
 * Copyright (c) 2014-2015, NVIDIA CORPORATION
 * Copyright (c) 2015, Nuno Subtil <subtil@gmail.com>
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the copyright holders nor the names of its
 *      contributors may be used to endorse or promote products derived from
 *      this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "from_nvbio/dna.h"

#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>

#include <lift/parallel.h>

#include "cigar.h"
#include "device_types.h"
#include "firepony_context.h"
#include "alignment_data_device.h"
#include "util.h"

namespace firepony {

// compute the length of a given cigar operator
struct cigar_op_len : public thrust::unary_function<const cigar_op&, uint32>
{
    CUDA_HOST_DEVICE uint32 operator() (const cigar_op& op) const
    {
        return op.len;
    }
};

template <target_system system>
struct compute_error_vectors : public lambda<system>
{
    LAMBDA_INHERIT_MEMBERS;

    pointer<system, uint8> snp_vector;
    pointer<system, uint8> ins_vector;
    pointer<system, uint8> del_vector;

    compute_error_vectors(firepony_context<system> ctx,
                          const alignment_batch_device<system> batch,
                          pointer<system, uint8> snp_vector,
                          pointer<system, uint8> ins_vector,
                          pointer<system, uint8> del_vector)
        : lambda<system>(ctx, batch),
          snp_vector(snp_vector),
          ins_vector(ins_vector),
          del_vector(del_vector)
    { }

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const bool negative_strand = batch.flags[read_index] & AlignmentFlags::REVERSE;

        auto reference = ctx.reference_db.get_sequence_data(batch.chromosome[read_index],
                                                            batch.alignment_start[read_index]);

        const auto read_window_clipped = ctx.cigar.read_window_clipped[read_index];
        const auto reference_window_clipped = ctx.cigar.reference_window_clipped[read_index];

        uint16 current_bp_idx = 0;
        uint16 num_errors = 0;

        // go through the cigar events looking for the event we're interested in
        for(uint32 event = idx.cigar_start; event < idx.cigar_start + idx.cigar_len; event++)
        {
            // figure out the cigar event range for this event
            const uint32 cigar_start = ctx.cigar.cigar_offsets[event];
            const uint32 cigar_end = ctx.cigar.cigar_offsets[event+1];

            switch(batch.cigars[event].op)
            {
            case cigar_op::OP_M:

                for(uint32 i = cigar_start; i < cigar_end; i++)
                {
                    // update the current read bp index
                    current_bp_idx = ctx.cigar.cigar_event_read_coordinates[i];
                    // load the read bp
                    const uint8 read_bp = batch.reads[idx.read_start + current_bp_idx];

                    // load the corresponding sequence bp
                    const uint32 reference_bp_idx = ctx.cigar.cigar_event_reference_coordinates[i];
                    const uint8 reference_bp = reference[reference_bp_idx];

                    if (reference_bp != read_bp)
                    {
                        snp_vector[idx.read_start + current_bp_idx] = 1;

                        // if we are inside the clipped read window, count this error
                        if (current_bp_idx >= read_window_clipped.x && current_bp_idx <= read_window_clipped.y)
                            num_errors++;
                    }
                }

                break;

            case cigar_event::I:
                // mark the read bp where an insertion begins
                current_bp_idx = ctx.cigar.cigar_event_read_coordinates[cigar_start];

                if (current_bp_idx >= read_window_clipped.x && current_bp_idx <= read_window_clipped.y)
                {
                    int off;

                    if (!negative_strand)
                    {
                        off = current_bp_idx - 1;
                    } else {
                        off = current_bp_idx + batch.cigars[event].len;
                    }

                    if (off >= 0 && off <= read_window_clipped.y)
                    {
                        ins_vector[idx.read_start + off] = 1;
                        num_errors++;
                    }
                }

                break;

            case cigar_event::D:
                // note: deletions do not exist in the read, so current_bp_idx is not updated here
                // also, because of this, we need to test against reference coordinates instead
                uint16 current_ref_idx = ctx.cigar.cigar_event_reference_coordinates[cigar_start];

                if (current_ref_idx >= reference_window_clipped.x && current_ref_idx <= reference_window_clipped.y)
                {
                    // mark the read bp where a deletion begins
                    if (!negative_strand)
                    {
                        del_vector[idx.read_start + current_bp_idx] = 1;
                        num_errors++;
                    } else {
                        uint16 off = current_bp_idx + 1;
                        if (off < idx.read_len)
                        {
                            del_vector[idx.read_start + off] = 1;
                            num_errors++;
                        }
                    }

                }

                break;
            }
        }

        ctx.cigar.num_errors[read_index] = num_errors;
    }
};

// expand cigar ops into temp storage
template <target_system system>
struct cigar_op_expand : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 op_index)
    {
        const cigar_op& op = batch.cigars[op_index];
        const uint32 out_base = ctx.cigar.cigar_offsets[op_index];

        uint8 *out = &ctx.temp_storage[0] + out_base;

        for(uint32 i = 0; i < op.len; i++)
        {
            switch(op.op)
            {
            case cigar_op::OP_M:
            case cigar_op::OP_MATCH:
            case cigar_op::OP_X:
                out[i] = cigar_event::M;
                break;

            case cigar_op::OP_I:
            case cigar_op::OP_N:
                out[i] = cigar_event::I;
                break;

            case cigar_op::OP_D:
            case cigar_op::OP_H:
            case cigar_op::OP_P:
                out[i] = cigar_event::D;
                break;

            case cigar_op::OP_S:
                out[i] = cigar_event::S;
                break;
            }
        }
    }
};

template <target_system system>
struct work_on_cigar : public lambda<system>
{
    LAMBDA_INHERIT;

#if 0
// initialize read windows
// note: this does not initialize the reference window, as it needs to be computed once all clipping has been done
template <target_system system>
struct read_window_init : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void read_window_init(const uint32 read_index)    
    {
        const CRQ_index idx = batch.crq_index(read_index);
        ctx.cigar.read_window_clipped[read_index] = make_ushort2(0, idx.read_len - 1);
    }
#if 0
};

// clips sequencing adapters from the reads
template <target_system system>
struct remove_adapters : public lambda<system>
{
    LAMBDA_INHERIT;
#endif

    CUDA_HOST_DEVICE bool hasWellDefinedFragmentSize(const uint32 read_index)
    {
        const auto flags = batch.flags[read_index];
        const auto inferred_insert_size = batch.inferred_insert_size[read_index];

        if (inferred_insert_size == 0)
            // no adaptors in reads with mates in another chromosome or unmapped pairs
            return false;

        if (!(flags & AlignmentFlags::PAIRED))
            // only reads that are paired can be adaptor trimmed
            return false;

        if ((flags & AlignmentFlags::UNMAP) ||
            (flags & AlignmentFlags::MATE_UNMAP))
            // only reads when both reads are mapped can be trimmed
            return false;

        const bool rev = flags & AlignmentFlags::REVERSE;
        const bool mate_rev = flags & AlignmentFlags::MATE_REVERSE;
        if (rev == mate_rev)
            // sanity check to ensure that read1 and read2 aren't on the same strand
            return false;

        if (flags & AlignmentFlags::REVERSE)
        {
            // we're on the negative strand, so our read runs right to left
            return batch.alignment_stop[read_index] > batch.mate_alignment_start[read_index];
        } else {
            // we're on the positive strand, so our mate should be to our right (his start + insert size should be past our start)
            return batch.alignment_start[read_index] <= batch.mate_alignment_start[read_index] + inferred_insert_size;
        }
    }

    static constexpr uint32 CANNOT_COMPUTE_ADAPTOR_BOUNDARY = 0xffffffff;

    CUDA_HOST_DEVICE uint32 getAdaptorBoundary(const uint32 read_index)
    {
        if (!hasWellDefinedFragmentSize(read_index))
        {
            return CANNOT_COMPUTE_ADAPTOR_BOUNDARY;
        }

        if (batch.flags[read_index] & AlignmentFlags::REVERSE)
        {
            return uint32(batch.mate_alignment_start[read_index] - 1);
        } else {
            const int insertSize = (batch.inferred_insert_size[read_index] < 0 ? -batch.inferred_insert_size[read_index] : batch.inferred_insert_size[read_index]);
            return uint32(batch.alignment_start[read_index] + insertSize + 1);
        }
    }

    enum AdaptorTail
    {
        left,
        right,
    };

    CUDA_HOST_DEVICE int getReadCoordinateForReferenceCoordinate(const uint32 read_index, uint32 ref_coord, AdaptorTail tail)
    {
        const CRQ_index idx = batch.crq_index(read_index);

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        const uint32 cigar_stop = ctx.cigar.cigar_offsets[idx.cigar_start + idx.cigar_len];

        for(uint32 ev = cigar_start; ev < cigar_stop; ev++)
        {
            if (batch.alignment_start[read_index] + ctx.cigar.cigar_event_reference_coordinates[ev] == ref_coord)
            {
                uint16 read_coord = ctx.cigar.cigar_event_read_coordinates[ev];

                if (read_coord == uint16(-1))
                {
                    // if there is no read coordinate, we must be in a deletion
                    // move forward/backward (depending on which tail we're locating) until we find a valid clipping point

                    if (tail == AdaptorTail::right)
                    {
                        while(ev < cigar_stop)
                        {
                            read_coord = ctx.cigar.cigar_event_read_coordinates[ev];

                            if (read_coord != uint16(-1))
                                break;

                            ev++;
                        }
                    } else {
                        while (ev >= cigar_start && ev > 0)
                        {
                            ev--;

                            read_coord = ctx.cigar.cigar_event_read_coordinates[ev];

                            if (read_coord != uint16(-1))
                                break;
                        }
                    }

                    // if we get here, then we failed to find a clipping coordinate
                    // this should not happen unless the read is malformed
                    if (read_coord == uint16(-1))
                        return -1;
                }

                return read_coord;
            }
        }

        return -1;
    }

    CUDA_HOST_DEVICE ushort2 hardClipByReferenceCoordinates(const uint32 read_index, int refStart, int refStop)
    {
        int start;
        int stop;

        if (refStart < 0)
        {
            start = 0;
            stop = getReadCoordinateForReferenceCoordinate(read_index, uint32(refStop), AdaptorTail::left);
        } else {
            start = getReadCoordinateForReferenceCoordinate(read_index, uint32(refStart), AdaptorTail::right);
            stop = ctx.cigar.read_window_clipped[read_index].y - ctx.cigar.read_window_clipped[read_index].x - 1;
        }

        return make_ushort2(start, stop);
    }

    CUDA_HOST_DEVICE void hardClipByReferenceCoordinates_LeftTail(const uint32 read_index, int refStop)
    {
        auto& read_window_clipped = ctx.cigar.read_window_clipped[read_index];

        ushort2 adapter = hardClipByReferenceCoordinates(read_index, -1, refStop);
        read_window_clipped.x = max<uint16>(read_window_clipped.x, adapter.y + 1);
    }

    CUDA_HOST_DEVICE void hardClipByReferenceCoordinates_RightTail(const uint32 read_index, int refStart)
    {
        auto& read_window_clipped = ctx.cigar.read_window_clipped[read_index];

        ushort2 adapter = hardClipByReferenceCoordinates(read_index, refStart, -1);
        read_window_clipped.y = min<uint16>(read_window_clipped.y, adapter.x - 1);
    }

    // this is essentially a copy of the compute_reference_window functor, except it uses the current read window (with indels)
    // we need to compute this early for adapter clipping, but the results will be out of date as soon as we're finished
    CUDA_HOST_DEVICE ushort2 get_current_reference_window(const uint32 read_index)
    {
        const auto& read_window_clipped = ctx.cigar.read_window_clipped[read_index];
        ushort2 reference_window_clipped;

        auto idx = batch.crq_index(read_index);
        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        const uint32 cigar_end = ctx.cigar.cigar_offsets[idx.cigar_start + idx.cigar_len];

        // do a linear search for the read offset
        // (this could be smarter, but it doesn't seem to matter)
        for(uint32 i = cigar_start; i < cigar_end; i++)
        {
            if (ctx.cigar.cigar_event_read_coordinates[i] == read_window_clipped.x)
            {
                while(ctx.cigar.cigar_event_reference_coordinates[i] == uint16(-1) &&
                        i < cigar_end)
                {
                    i++;
                }

                if (i == cigar_end)
                {
                    // should never happen
                    reference_window_clipped = make_ushort2(uint16(-1), uint16(-1));
                    return reference_window_clipped;
                }

                reference_window_clipped.x = ctx.cigar.cigar_event_reference_coordinates[i];
                break;
            }
        }

        for(uint32 i = cigar_end - 1; i >= cigar_start; i--)
        {
            if (ctx.cigar.cigar_event_read_coordinates[i] == read_window_clipped.y)
            {
                while(ctx.cigar.cigar_event_reference_coordinates[i] == uint16(-1) &&
                        i > cigar_start)
                {
                    i--;
                }

                if (i == cigar_start)
                {
                    // should never happen
                    reference_window_clipped = make_ushort2(uint16(-1), uint16(-1));
                    return reference_window_clipped;
                }

                reference_window_clipped.y = ctx.cigar.cigar_event_reference_coordinates[i];
                break;
            }
        }

        return reference_window_clipped;
    }

#if 0
    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void remove_adapters(const uint32 read_index)
    {
        uint32 adaptorBoundary = getAdaptorBoundary(read_index);

        if (adaptorBoundary == CANNOT_COMPUTE_ADAPTOR_BOUNDARY)
            return;

        const auto reference_window = get_current_reference_window(read_index);

        if (adaptorBoundary < batch.alignment_start[read_index] + reference_window.x ||
            adaptorBoundary > batch.alignment_start[read_index] + reference_window.y)
        {
            return;
        }

        if (batch.flags[read_index] & AlignmentFlags::REVERSE)
        {
            hardClipByReferenceCoordinates_LeftTail(read_index, adaptorBoundary);
        } else {
            hardClipByReferenceCoordinates_RightTail(read_index, adaptorBoundary);
        }
    }
#if 0
};

// remove soft-clip regions from the active read window
template <target_system system>
struct remove_soft_clips : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void remove_soft_clips(const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);

        auto read_window_clipped = ctx.cigar.read_window_clipped[read_index];

        uint32 cigar_index = idx.cigar_start;
        uint32 read_offset = 0;

        // note: we assume that the leading/trailing clip regions have been validated by the read filtering stage!

        // iterate forward through the leading clip region
        while(cigar_index < idx.cigar_start + idx.cigar_len &&
              read_offset < idx.read_len)
        {
            const auto& op = batch.cigars[cigar_index];

            if (op.op == cigar_op::OP_H || op.op == cigar_op::OP_S)
            {
                if (read_offset + op.len > read_window_clipped.x)
                {
                    read_window_clipped.x = min<uint16>(read_offset + op.len, read_window_clipped.y);
                }

                read_offset += op.len;
                cigar_index++;
            } else {
                break;
            }
        }

        // iterate backward through the trailing clip region
        cigar_index = idx.cigar_start + idx.cigar_len - 1;
        read_offset = idx.read_len - 1;

        while(cigar_index >= idx.cigar_start &&
              read_offset > read_window_clipped.x)
        {
            const auto& op = batch.cigars[cigar_index];

            if (op.op == cigar_op::OP_H || op.op == cigar_op::OP_S)
            {
                if (read_offset - op.len < read_window_clipped.y)
                {
                    read_window_clipped.y = max<uint16>(read_offset - op.len, read_window_clipped.x);
                }

                read_offset -= op.len;
                cigar_index--;
            } else {
                break;
            }
        }

        ctx.cigar.read_window_clipped[read_index] = read_window_clipped;
    }
#if 0
};

// compute clipped read window without leading/trailing insertions
template <target_system system>
struct compute_no_insertions_window : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void compute_no_insertions_window(const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const auto& read_window_clipped = ctx.cigar.read_window_clipped[read_index];
        auto& read_window_clipped_no_insertions = ctx.cigar.read_window_clipped_no_insertions[read_index];

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        const uint32 cigar_end = ctx.cigar.cigar_offsets[idx.cigar_start + idx.cigar_len];

        uint32 ev;

        // iterate forward at the start
        ev = cigar_start;
        while(ev < cigar_end)
        {
            const uint16 read_offset = ctx.cigar.cigar_event_read_coordinates[ev];
            // skip bases without a read offset and bases behind the current clipping window
            if (read_offset == uint16(-1) || read_offset < read_window_clipped.x)
            {
                ev++;
                continue;
            }

            if (ctx.cigar.cigar_events[ev] == cigar_event::I)
            {
                ev++;
                continue;
            }

            read_window_clipped_no_insertions.x = read_offset;
            break;
        }

        // iterate backwards from the end
        ev = cigar_end - 1;
        while(ev >= cigar_start && ev < cigar_end)
        {
            const uint16 read_offset = ctx.cigar.cigar_event_read_coordinates[ev];
            // skip bases without a read offset and bases beyond the current clipping window
            if (read_offset == uint16(-1) || read_offset > read_window_clipped.y)
            {
                ev--;
                continue;
            }

            if (ctx.cigar.cigar_events[ev] == cigar_event::I)
            {
                ev--;
                continue;
            }

            read_window_clipped_no_insertions.y = read_offset;
            break;
        }
    }
#if 0
};

template <target_system system>
struct compute_reference_window : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void compute_reference_window(const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);

        const auto& read_window_clipped_no_insertions = ctx.cigar.read_window_clipped_no_insertions[read_index];
        auto& reference_window_clipped = ctx.cigar.reference_window_clipped[read_index];

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        const uint32 cigar_end = ctx.cigar.cigar_offsets[idx.cigar_start + idx.cigar_len];

        // do a linear search for the read offset
        // (this could be smarter, but it doesn't seem to matter)
        for(uint32 i = cigar_start; i < cigar_end; i++)
        {
            if (ctx.cigar.cigar_event_read_coordinates[i] == read_window_clipped_no_insertions.x)
            {
                while(ctx.cigar.cigar_event_reference_coordinates[i] == uint16(-1) &&
                        i < cigar_end)
                {
                    i++;
                }

                if (i == cigar_end)
                {
                    // should never happen
                    reference_window_clipped = make_ushort2(uint16(-1), uint16(-1));
                    return;
                }

                reference_window_clipped.x = ctx.cigar.cigar_event_reference_coordinates[i];
                break;
            }
        }

        for(uint32 i = cigar_end - 1; i >= cigar_start; i--)
        {
            if (ctx.cigar.cigar_event_read_coordinates[i] == read_window_clipped_no_insertions.y)
            {
                while(ctx.cigar.cigar_event_reference_coordinates[i] == uint16(-1) &&
                        i > cigar_start)
                {
                    i--;
                }

                if (i == cigar_start)
                {
                    // should never happen
                    reference_window_clipped = make_ushort2(uint16(-1), uint16(-1));
                    return;
                }

                reference_window_clipped.y = ctx.cigar.cigar_event_reference_coordinates[i];
                break;
            }
        }
    }
#if 0
};
#endif

// expand cigar coordinates for a read
// xxxnsubtil: this is very similar to compute_alignment_window, should merge
#if 0
template <target_system system>
struct cigar_coordinates_expand : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
#endif
    LIFT_HOST_DEVICE void cigar_coordinates_expand(const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const cigar_op *cigar = &batch.cigars[idx.cigar_start];

        uint32 base = ctx.cigar.cigar_offsets[idx.cigar_start];
        uint32 *output_read_index = &ctx.cigar.cigar_event_read_index[base];
        uint16 *output_read_coordinates = &ctx.cigar.cigar_event_read_coordinates[base];
        uint16 *output_reference_coordinates = &ctx.cigar.cigar_event_reference_coordinates[base];

        uint16 read_offset = 0;
        uint16 reference_offset = 0;

        for(uint32 c = 0; c < idx.cigar_len; c++)
        {
            switch(cigar[c].op)
            {
            case cigar_op::OP_M:
            case cigar_op::OP_MATCH:
            case cigar_op::OP_X:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    *output_read_index++ = read_index;
                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = reference_offset;

                    read_offset++;
                    reference_offset++;
                }

                break;

            case cigar_op::OP_S:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    *output_read_index++ = read_index;
                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = uint16(-1);

                    read_offset++;
                }

                break;

            case cigar_op::OP_N: // xxxnsubtil: N is really not supported and shouldn't be here
            case cigar_op::OP_I:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    *output_read_index++ = read_index;
                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = uint16(-1);

                    read_offset++;
                }

                break;

            case cigar_op::OP_D:
            case cigar_op::OP_H:
            case cigar_op::OP_P: // xxxnsubtil: not sure how to handle P
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    *output_read_index++ = read_index;
                    *output_read_coordinates++ = uint16(-1);
                    *output_reference_coordinates++ = reference_offset;

                    reference_offset++;
                }
            }
        }
    }
#if 0
};
#endif

    LIFT_HOST_DEVICE void operator() (const uint32 read_index)
    {
        cigar_coordinates_expand(read_index);
        read_window_init(read_index);
        remove_adapters(read_index);
        remove_soft_clips(read_index);
        compute_no_insertions_window(read_index);
        compute_reference_window(read_index);
    }
};

#ifdef CUDA_DEBUG
// debug aid: sanity check that the expanded cigar events match what we expect
template <target_system system>
struct sanity_check_cigar_events : public lambda<system>
{
    LAMBDA_INHERIT;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const cigar_op *cigar = &batch.cigars[idx.cigar_start];

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        uint32 cigar_event_idx = 0;

        for(uint32 c = 0; c < idx.cigar_len; c++)
        {
            for(uint32 i = 0; i < cigar[c].len; i++)
            {
                switch(cigar[c].op)
                {
                case cigar_op::OP_M:
                case cigar_op::OP_MATCH:
                case cigar_op::OP_X:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::M)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected M, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_N: // xxxnsubtil: N is really not supported and shouldn't be here
                case cigar_op::OP_I:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::I)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected I, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_D:
                case cigar_op::OP_H:
                case cigar_op::OP_P: // xxxnsubtil: not sure how to handle P
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::D)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected D, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_S:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::S)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected S, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;
                }
            }
        }
    }
};
#endif

template <target_system system>
void expand_cigars(firepony_context<system>& context, const alignment_batch<system>& batch)
{
    auto& ctx = context.cigar;

    // compute the offsets of each expanded cigar op
    // xxxnsubtil: we ignore the active read list here, so we do unnecessary work
    // might want to revisit this
    ctx.cigar_offsets.resize(batch.device.cigars.size() + 1);

    // mark the first offset as 0
    thrust::fill_n(lift::backend_policy<system>::execution_policy(), ctx.cigar_offsets.begin(), 1, 0);
    // do an inclusive scan to compute all offsets + the total size
    parallel<system>::inclusive_scan(thrust::make_transform_iterator(batch.device.cigars.begin(), cigar_op_len()),
                                     batch.device.cigars.size(),
                                     ctx.cigar_offsets.begin() + 1,
                                     thrust::plus<uint32>());

    // read back the last element, which contains the size of the buffer required
    uint32 expanded_cigar_len = ctx.cigar_offsets.peek(batch.device.cigars.size());

    // make sure we have enough room for the expanded cigars
    // note: temporary storage must be padded to a multiple of the word size, since we'll pack whole words at a time
    pack_prepare_storage_2bit(context.temp_storage, expanded_cigar_len);
    ctx.cigar_events.resize(expanded_cigar_len);

    ctx.cigar_event_read_index.resize(expanded_cigar_len);
    ctx.cigar_event_reference_coordinates.resize(expanded_cigar_len);
    ctx.cigar_event_read_coordinates.resize(expanded_cigar_len);

    ctx.read_window_clipped.resize(batch.device.num_reads);
    ctx.read_window_clipped_no_insertions.resize(batch.device.num_reads);
    ctx.reference_window_clipped.resize(batch.device.num_reads);

    ctx.is_snp.resize(batch.device.reads.size());
    ctx.is_insertion.resize(batch.device.reads.size());
    ctx.is_deletion.resize(batch.device.reads.size());
    ctx.num_errors.resize(batch.device.num_reads);

    // initialize num_errors to zero
    thrust::fill(lift::backend_policy<system>::execution_policy(), ctx.num_errors.begin(), ctx.num_errors.end(), 0);

    // cigar_events_read_index is initialized to -1; this means that all reads are considered inactive
    // it will be filled in during cigar coordinate expansion to mark active reads
    thrust::fill(lift::backend_policy<system>::execution_policy(), ctx.cigar_event_read_index.begin(), ctx.cigar_event_read_index.end(), uint32(-1));

    // expand the cigar ops into temp storage (xxxnsubtil: same as above, active read list is ignored)
    parallel<system>::for_each(thrust::make_counting_iterator(0),
                               thrust::make_counting_iterator(0) + batch.device.cigars.size(),
                               cigar_op_expand<system>(context, batch.device));

    // pack the cigar into a 2-bit vector
    pack_to_2bit(ctx.cigar_events, context.temp_storage);

#ifdef CUDA_DEBUG
    parallel<system>::for_each(firepony_context.active_read_list.begin(),
                               firepony_context.active_read_list.end(),
                               sanity_check_cigar_events<system>(firepony_context, batch.device));
#endif

#if 0
    // now expand the coordinates per read
    // this avoids having to deal with boundary conditions within reads
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               cigar_coordinates_expand<system>(context, batch.device));

    // initialize read windows
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               read_window_init<system>(context, batch.device));

    // remove sequencing adapters
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               remove_adapters<system>(context, batch.device));

    // remove soft clip regions
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               remove_soft_clips<system>(context, batch.device));

    // compute the no insertions window based on the clipping window
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               compute_no_insertions_window<system>(context, batch.device));

    // finally, compute the reference window (using the no insertions window)
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               compute_reference_window<system>(context, batch.device));
#else
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               work_on_cigar<system>(context, batch.device));
#endif

    // compute the error bit vectors
    // this also counts the number of errors in each read
    // note: we compute the error bit vectors into uint8 then pack these into 1-bit-per-bp vectors
    // this is to avoid RMW hazards across threads, as the number of symbols per word won't match that of the read vectors themselves
    allocation<system, uint8>& snp_error = context.temp_storage;
    allocation<system, uint8>& ins_error = context.temp_u8;
    scoped_allocation<system, uint8> del_error;

    // set up the temp storage for packing into 1bit
    size_t len = batch.device.reads.size();
    pack_prepare_storage_1bit(snp_error, len);
    pack_prepare_storage_1bit(ins_error, len);
    pack_prepare_storage_1bit(del_error, len);

    // initialize temp storage to zero
    thrust::fill(lift::backend_policy<system>::execution_policy(), snp_error.begin(), snp_error.end(), 0);
    thrust::fill(lift::backend_policy<system>::execution_policy(), ins_error.begin(), ins_error.end(), 0);
    thrust::fill(lift::backend_policy<system>::execution_policy(), del_error.begin(), del_error.end(), 0);

    // compute the error bit vectors into temp storage
    parallel<system>::for_each(context.active_read_list.begin(),
                               context.active_read_list.end(),
                               compute_error_vectors<system>(context, batch.device,
                                                             snp_error, ins_error, del_error));

    // now pack the temp storage into the 1-bit vectors
    pack_to_1bit(context.cigar.is_snp, snp_error);
    pack_to_1bit(context.cigar.is_insertion, ins_error);
    pack_to_1bit(context.cigar.is_deletion, del_error);
}
INSTANTIATE(expand_cigars);

template <target_system system>
void debug_cigar(firepony_context<system>& context, const alignment_batch<system>& batch, int read_index)
{
    const auto& h_batch = *batch.host;

    const CRQ_index idx = h_batch.crq_index(read_index);
    const auto& ctx = context.cigar;

    ushort2 read_window_clipped = ctx.read_window_clipped[read_index];

    fprintf(stderr, "  cigar info:\n");

    fprintf(stderr, "    cigar                       = [");
    for(uint32 i = idx.cigar_start; i < idx.cigar_start + idx.cigar_len; i++)
    {
        cigar_op op = h_batch.cigars[i];
        fprintf(stderr, "%d%c", op.len, op.ascii_op());
    }
    fprintf(stderr, "]\n");

    uint32 cigar_start = ctx.cigar_offsets[idx.cigar_start];
    uint32 cigar_end = ctx.cigar_offsets[idx.cigar_start + idx.cigar_len];
    fprintf(stderr, "    offset range                = [% 3d, % 3d]\n", cigar_start, cigar_end);

    fprintf(stderr, "                                    ");
    for(uint32 i = 0; i < cigar_end - cigar_start; i++)
    {
        fprintf(stderr, "% 4d ", i);
    }
    fprintf(stderr, "\n");

    fprintf(stderr, "    event list                  = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        fprintf(stderr, "   %c ", cigar_event::ascii(ctx.cigar_events[i]));
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    event idx -> read coords    = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        fprintf(stderr, "% 4d ", (int16) ctx.cigar_event_read_coordinates[i]);
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "             ... clipped        = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        int16 coord = ctx.cigar_event_read_coordinates[i] - read_window_clipped.x;

        if (coord >= 0)
        {
            fprintf(stderr, "% 4d ", coord);
        } else {
            fprintf(stderr, "   - ");
        }
    }
    fprintf(stderr, "]\n");


    fprintf(stderr, "    event reference coordinates = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        fprintf(stderr, "% 4d ", (int16) ctx.cigar_event_reference_coordinates[i]);
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    is snp                      = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            fprintf(stderr, "%s", (uint8) ctx.is_snp[idx.read_start + read_bp_idx] ? "   1 " : "   . ");
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    is insertion                = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            fprintf(stderr, "%s", (uint8) ctx.is_insertion[idx.read_start + read_bp_idx] ? "   1 " : "   . ");
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    is deletion                 = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            fprintf(stderr, "%s", (uint8) ctx.is_deletion[idx.read_start + read_bp_idx] ? "   1 " : "   . ");
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "\n");

    fprintf(stderr, "    skip list                   = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 bp_offset = ctx.cigar_event_read_coordinates[i];
        if (bp_offset == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            if (context.active_location_list[idx.read_start + bp_offset] == 0)
                fprintf(stderr, "% 4d ", 1);
            else
                fprintf(stderr, "   . ");
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "\n");

    fprintf(stderr, "    fractional snp error        = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 bp_offset = ctx.cigar_event_read_coordinates[i];
        if (bp_offset == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            double err = context.fractional_error.snp_errors[idx.qual_start + bp_offset];
            if (err == 0.0)
                fprintf(stderr, "   . ");
            else
                fprintf(stderr, " %.2f", err);
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "           ... ins error        = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 bp_offset = ctx.cigar_event_read_coordinates[i];
        if (bp_offset == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            double err = context.fractional_error.insertion_errors[idx.qual_start + bp_offset];
            if (err == 0.0)
                fprintf(stderr, "   . ");
            else
                fprintf(stderr, " %.2f", err);
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "           ... del error        = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 bp_offset = ctx.cigar_event_read_coordinates[i];
        if (bp_offset == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            double err = context.fractional_error.deletion_errors[idx.qual_start + bp_offset];
            if (err == 0.0)
                fprintf(stderr, "   . ");
            else
                fprintf(stderr, " %.2f", err);
        }
    }
    fprintf(stderr, "]\n");

    const auto& reference_db = ((const sequence_database_storage<system>) context.reference_db);
    const auto& reference = reference_db.get_sequence_data(h_batch.chromosome[read_index],
                                                           h_batch.alignment_start[read_index]);

    fprintf(stderr, "    reference sequence data     = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 ref_bp = ctx.cigar_event_reference_coordinates[i];
        fprintf(stderr, "   %c ", ref_bp == uint16(-1) ? '-' : from_nvbio::iupac16_to_char(reference[ref_bp]));
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    read sequence data          = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];
        char base;

        if (read_bp == uint16(-1))
        {
            base = '-';
        } else {
            base = from_nvbio::iupac16_to_char(h_batch.reads[idx.read_start + read_bp]);
            if (ctx.cigar_events[i] == cigar_event::S)
            {
                // display soft-clipped bases in lowercase
                base = tolower(base);
            }
        }

        fprintf(stderr, "   %c ", base);
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    read quality data           = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];

        if (read_bp == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            fprintf(stderr, "% 4d ", h_batch.qualities[idx.qual_start + read_bp]);
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    ... in ascii                = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];

        if (read_bp == uint16(-1))
        {
            fprintf(stderr, "   - ");
        } else {
            fprintf(stderr, "   %c ", h_batch.qualities[idx.qual_start + read_bp] + '!');
        }
    }
    fprintf(stderr, "]\n");

    fprintf(stderr, "    clipped read window         = [ % 3d, % 3d ]\n", read_window_clipped.x, read_window_clipped.y);

    ushort2 read_window_clipped_no_insertions = ctx.read_window_clipped_no_insertions[read_index];
    fprintf(stderr, "    ... lead/trail insertions   = [ % 3d, % 3d ]\n",
                read_window_clipped_no_insertions.x, read_window_clipped_no_insertions.y);

    ushort2 reference_window_clipped = ctx.reference_window_clipped[read_index];
    fprintf(stderr, "    clipped reference window    = [ % 3d, % 3d ]\n",
                reference_window_clipped.x, reference_window_clipped.y);

    uint16 err = ctx.num_errors[read_index];
    fprintf(stderr, "    errors in clipped region    = [ % 3d ]\n", err);

    fprintf(stderr, "\n");
}
INSTANTIATE(debug_cigar);

} // namespace firepony

